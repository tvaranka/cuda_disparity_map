#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <string>


__global__
void disparity_map(unsigned char* img0, unsigned char* img1, unsigned char* disp_img0,
				   unsigned char* disp_img1, unsigned width, unsigned height){
	const unsigned B = 5;
	const unsigned filter_size = B * B;
	const unsigned d_max = 50;
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	
	int i, j;
	float current_max0 = -1, current_max1 = -1;
    int best_d0 = 0, best_d1 = 0;
    float zncc0, zncc1;

    if ((x < width - 1 - B - d_max) && (y < height - 1 - B)
        && (x >= d_max)){
	    for (int d = 0; d <= d_max; d++){
	        float img0_mean0 = 0, img0_mean1 = 0, img1_mean0 = 0, img1_mean1 = 0;

	        for (i = 0; i < B; i++){
	            for (j = 0; j < B; j++){
	            	img0_mean0 += img0[(y + i) * width + x + j];
	                img0_mean1 += img0[(y + i) * width + x + j + d];
	                img1_mean0 += img1[(y + i) * width + x + j - d];
	                img1_mean1 += img1[(y + i) * width + x + j];
	            }
	        }

	        img0_mean0 /= filter_size; img1_mean0 /= filter_size;
	        img0_mean1 /= filter_size; img1_mean1 /= filter_size;
	        
	        float top0 = 0, top1 = 0, bot00 = 0, bot01 = 0, bot10 = 0, bot11 = 0;
	        for (i = 0; i < B; i++){
	            for (j = 0; j < B; j++){
	                top0 += (img0[(y + i) * width + x + j] - img0_mean0)
	                        * (img1[(y + i) * width + x + j - d] - img1_mean0);
	                bot00 += powf((img0[(y + i) * width + x + j] - img0_mean0), 2);
	                bot10 += powf((img1[(y + i) * width + x + j - d] - img1_mean0), 2);

	                top1 += (img0[(y + i) * width + x + j + d] - img0_mean1)
	                        * (img1[(y + i) * width + x + j] - img1_mean1);
	                bot01 += powf((img0[(y + i) * width + x + j + d] - img0_mean1), 2);
	                bot11 += powf((img1[(y + i) * width + x + j] - img1_mean1), 2);
	            }
	        }
	        zncc0 = top0 / (sqrtf(bot00 * bot10));
	        zncc1 = top1 / (sqrtf(bot01 * bot11));
	        if (zncc0 > current_max0){
	            current_max0 = zncc0;
	            best_d0 = d;
	        }
	            
	        if (zncc1 > current_max1){
	            current_max1 = zncc1;
	            best_d1 = d;
	        }
	    }
	    disp_img0[y * width + x] = (int)(best_d0 * (255.0 / d_max));
	    disp_img1[y * width + x] = (int)(best_d1 * (255.0 / d_max));
	}
	else{
		disp_img0[y * width + x] = 0;
	    disp_img1[y * width + x] = 0;
	}
}

__global__
void threshold(unsigned char* disp_img0, unsigned char* disp_img1,
	unsigned width, unsigned height, int threshold){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int diff;
	int id = y * width + x;

	diff = abs(disp_img0[id] - disp_img1[id]);
	disp_img0[id] = (diff < threshold) * disp_img0[id];
	//occlusion filling
	int range = 0;
	if (disp_img0[y * width + x] == 0){
		for (int i = 0; i < range; i++){
			for (int j = 0; j < range; j++){
				if ((y + i < height) && (x + j < width)){
					if (disp_img0[(y + i) * width + x + j] != 0){
						disp_img1[id] = disp_img0[(y + i) * width + x + j];
						i = j = 10000;
						break;
					}
				}
			}
		}
	}
	__syncthreads();
	disp_img0[id] = ((id > 0) && (id < width * height)) ? (disp_img1[id] + disp_img1[id + 1]
					 + disp_img1[id - 1]) / 3 : 0;
}


int main() {
	
	cv::Mat img0 = cv::imread("../../data/im0small.png", cv::IMREAD_GRAYSCALE);
	cv::Mat img1 = cv::imread("../../data/im1small.png", cv::IMREAD_GRAYSCALE);
	if (img0.empty() || img1.empty()) {
		std::cout << "Failed loading image" << std::endl;
		std::cin.get();
		return -1;
	}
	
	unsigned char *d_img0, *d_img1, *d_disp_img0, *d_disp_img1;
	const int height = img0.rows;
	const int width = img0.cols;
	cv::Mat res(height, width, CV_8UC1);
	const dim3 block(32, 32);
	const dim3 grid(width / block.x, height / block.y);
	const size_t tsize = width * height * sizeof(unsigned char);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc<unsigned char>(&d_img0, tsize);
	hipMalloc<unsigned char>(&d_img1, tsize);
	hipMalloc<unsigned char>(&d_disp_img0, tsize);
	hipMalloc<unsigned char>(&d_disp_img1, tsize);
	hipMemcpy(d_img0, img0.ptr(), tsize, hipMemcpyHostToDevice);
	hipMemcpy(d_img1, img1.ptr(), tsize, hipMemcpyHostToDevice);

	hipEventRecord(start);
	disparity_map<<<grid, block>>>(d_img0, d_img1, d_disp_img0, d_disp_img1, width, height);
	hipDeviceSynchronize();
    //threshold<<<grid, block>>>(d_disp_img0, d_disp_img1, width, height, 12);
    //hipDeviceSynchronize();
	hipEventRecord(stop);

	hipMemcpy(res.ptr(), d_disp_img0, tsize, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	std::cout << "Time(ms): " << milliseconds << std::endl;
	std::cout << res.size() << std::endl;
	//cv::imwrite("../depth_full_size.jpg", res);
	cv::namedWindow("Window name", cv::WINDOW_NORMAL);
	cv::imshow("Window name", res);
	cv::waitKey(0);
	hipFree(d_img0);
	hipFree(d_img1);
	hipFree(d_disp_img0);
	hipFree(d_disp_img1);
	//cv::destroyWindow(window_name);
	return 0;
}